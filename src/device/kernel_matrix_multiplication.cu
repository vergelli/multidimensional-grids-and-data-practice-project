#include <hip/hip_runtime.h>

//TODO: revisar kernel
__global__ void matrixMulKernel(float* A, float* B, float* C, int rowsA, int colsA, int colsB) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rowsA && col < colsB) {
        float value = 0.0f;
        for (int k = 0; k < colsA; ++k) {
            value += A[row * colsA + k] * B[k * colsB + col];
        }
        C[row * colsB + col] = value;
    }
}
